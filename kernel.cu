#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "Chroma.h"

__global__ void chroma_CUDA(unsigned char* Image, unsigned char* Input_Image2, int Channels);

void Image_Chroma_CUDA(unsigned char* Input_Image, unsigned char* Input_Image2, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;
	unsigned char* Dev_Input_Image2 = NULL;
	//allocate the memory in gpu
	hipMalloc((void**)& Dev_Input_Image, Height * Width * Channels);
	hipMalloc((void**)& Dev_Input_Image2, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);
	hipMemcpy(Dev_Input_Image2, Input_Image2, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	chroma_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Dev_Input_Image2, Channels);

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free gpu mempry
	hipFree(Dev_Input_Image);
}

__global__ void chroma_CUDA(unsigned char* Image, unsigned char* Image2, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		if (Image[idx + i + 1] == 255)
		{
			Image[idx + i] = Image2[idx + i];
			Image[idx + i + 1] = Image2[idx + i + 1];
			Image[idx + i + 2] = Image2[idx + i + 2];
		}

	}
}